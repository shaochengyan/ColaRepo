#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <torch/extension.h>
#include <torch/library.h>

using namespace at;


int64_t integer_round(int64_t num, int64_t denom){
  return (num + denom - 1) / denom;
}


template<class T>
__global__ void add_one_kernel(const T *const input, T *const output, const int64_t N){
  // Grid-strided loop
  for(int i=blockDim.x*blockIdx.x+threadIdx.x;i<N;i+=blockDim.x*gridDim.x){
    output[i] = input[i] + 1;
  }
}


///Adds one to each element of a tensor
Tensor add_one(const Tensor &input){
  auto output = torch::zeros_like(input);

  // Common values:
  // AT_DISPATCH_INDEX_TYPES
  // AT_DISPATCH_FLOATING_TYPES
  // AT_DISPATCH_INTEGRAL_TYPES
  AT_DISPATCH_ALL_TYPES(
    input.scalar_type(), "add_one_cuda", [&](){
      const auto block_size = 128;
      const auto num_blocks = std::min(65535L, integer_round(input.numel(), block_size));
      add_one_kernel<<<num_blocks, block_size>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        input.numel()
      );
      // Always test your kernel launches
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );

  return output;
}


///Note that we can have multiple implementations spread across multiple files, though there should only be one `def`
TORCH_LIBRARY(pytorch_cmake_example, m) {
  m.def("add_one(Tensor input) -> Tensor");
  m.impl("add_one", c10::DispatchKey::CUDA, TORCH_FN(add_one));
  //c10::DispatchKey::CPU is also an option
}