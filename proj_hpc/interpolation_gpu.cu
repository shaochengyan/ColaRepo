#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include "interpolation.h"


// template <typename scalar_t>
// __global__ void trilinear_fw_kernel(
//     const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
//     const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
//     torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
// ){
//     const int n = blockIdx.x * blockDim.x + threadIdx.x;
//     const int f = blockIdx.y * blockDim.y + threadIdx.y;

//     if (n>=feats.size(0) || f>=feats.size(2)) return;

//     // point -1~1
//     const scalar_t u = (points[n][0]+1)/2;
//     const scalar_t v = (points[n][1]+1)/2;
//     const scalar_t w = (points[n][2]+1)/2;
    
//     const scalar_t a = (1-v)*(1-w);
//     const scalar_t b = (1-v)*w;
//     const scalar_t c = v*(1-w);
//     const scalar_t d = 1-a-b-c;
//     feat_interp[n][f] = (1-u)*(a*feats[n][0][f] +
//                                b*feats[n][1][f] +
//                                c*feats[n][2][f] +
//                                d*feats[n][3][f]) + 
//                             u*(a*feats[n][4][f] +
//                                b*feats[n][5][f] +
//                                c*feats[n][6][f] +
//                                d*feats[n][7][f]);
// }

template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    // 写出对应的参数即可
    // input
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    // output
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
) {
    // 每个线程对应 第 i 个各自 的第 j 个 维度

    // step1: 固定的下标计算
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    const int f = blockDim.y * blockIdx.y + threadIdx.y;

    // step2: 排除不必要的 thread
    // TODO: 绘制图
    if ( n >= feats.size(0) || f >= feats.size(2) ) return;

    // 计算第 n 个点 的 第 f 个维度的特征 ( 8 个点的线性组合)
    // TODO: 四点现象组合 和 8点线性组合的公式
    // point \in (-1, 1) -> ratio: (x - (-1)) / (1 - (-1)
    const scalar_t u = (points[n][0] + 1) / 2;
    const scalar_t v = (points[n][1] + 1) / 2;
    const scalar_t w = (points[n][2] + 1) / 2;

    const scalar_t a = (1 - v) * (1 - w);
    const scalar_t b = (1 - v) * w;
    const scalar_t c = v * (1 - w);
    const scalar_t d = 1 - a - b - c;
    feat_interp[n][f] = (1 - u) * (
        a * feats[n][0][f] + 
        b * feats[n][1][f] + 
        c * feats[n][2][f] + 
        d * feats[n][3][f] 
    ) + 
    u * (
        a * feats[n][4][f] + 
        b * feats[n][5][f] + 
        c * feats[n][6][f] + 
        d * feats[n][7][f] 
    );

}



// template <typename scalar_t>
// __global__ void trilinear_fw_kernel(
//     // 写出对应的参数即可
//     // input
//     const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
//     const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
//     // output
//     torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
// ) {
//     // 每个线程对应 第 i 个各自 的第 j 个 维度

//     // step1: 固定的下标计算
//     const int n = blockDim.x * blockIdx.x + threadIdx.x;
//     const int f = blockDim.y * blockIdx.y + threadIdx.y;

//     // step2: 排除不必要的 thread
//     // TODO: 绘制图
//     if ( n >= feats.size(0) || f >= feats.size(2) ) return;

//     // 计算第 n 个点 的 第 f 个维度的特征 ( 8 个点的线性组合)
//     // TODO: 四点现象组合 和 8点线性组合的公式
//     // point \in (-1, 1) -> ratio: (x - (-1)) / (1 - (-1)
//     const scalar_t u = (points[n][0] + 1) / 2;
//     const scalar_t v = (points[n][1] + 1) / 2;
//     const scalar_t w = (points[n][2] + 1) / 2;

//     const scalar_t a = (1 - v) * (1 - w);
//     const scalar_t b = (1 - v) * w;
//     const scalar_t c = v * (1 - w);
//     const scalar_t d = 1 - a - b - c;
//     feat_interp[n][f] = (1 - u) * (
//         a * feats[n][0][f] + 
//         b * feats[n][1][f] + 
//         c * feats[n][2][f] + 
//         d * feats[n][3][f] 
//     ) + 
//     u * (
//         a * feats[n][4][f] + 
//         b * feats[n][5][f] + 
//         c * feats[n][6][f] + 
//         d * feats[n][7][f] 
//     );

// }


template <typename scalar_t>
__global__ void trilinear_bw_kernel(
    // 写出对应的参数即可
    // input
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dfeats_interp,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    // output
    torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> dL_dfeats
) {
    // 目的就是: 一个一个将 dL_dfeats 的值算出来! 即可!
    // 每个线程对应 第 i 个各自 的第 j 个 维度

    // step1: 固定的下标计算
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    const int f = blockDim.y * blockIdx.y + threadIdx.y;

    // step2: 排除不必要的 thread
    // TODO: 绘制图
    if ( n >= feats.size(0) || f >= feats.size(2) ) return;

    // 计算第 n 个点 的 第 f 个维度的特征 ( 8 个点的线性组合)
    // TODO: 四点现象组合 和 8点线性组合的公式
    // point \in (-1, 1) -> ratio: (x - (-1)) / (1 - (-1)
    const scalar_t u = (points[n][0] + 1) / 2;
    const scalar_t v = (points[n][1] + 1) / 2;
    const scalar_t w = (points[n][2] + 1) / 2;

    const scalar_t a = (1 - v) * (1 - w);
    const scalar_t b = (1 - v) * w;
    const scalar_t c = v * (1 - w);
    const scalar_t d = 1 - a - b - c;

    // 根据偏微分公式计算出偏微分的结果
    dL_dfeats[n][0][f] = (1 - u) * a * dL_dfeats_interp[n][f];
    dL_dfeats[n][1][f] = (1 - u) * b * dL_dfeats_interp[n][f];
    dL_dfeats[n][2][f] = (1 - u) * c * dL_dfeats_interp[n][f];
    dL_dfeats[n][3][f] = (1 - u) * d * dL_dfeats_interp[n][f];
    dL_dfeats[n][4][f] = (u) * a * dL_dfeats_interp[n][f];
    dL_dfeats[n][5][f] = (u) * b * dL_dfeats_interp[n][f];
    dL_dfeats[n][6][f] = (u) * c * dL_dfeats_interp[n][f];
    dL_dfeats[n][7][f] = (u) * d * dL_dfeats_interp[n][f];

}

torch::Tensor trilinear_fw_cu(
    torch::Tensor feats, 
    torch::Tensor points 
) {
    // 需要有一个 output, first 定义一个空的, 然后不断填进去,初始先明确其形状.
    const int N = feats.size(0), F = feats.size(2);  // shape[0]

    // torch.zeros(N, F, dtype=torch.float32, device="cuda:0")
    // dtype, device all store in tensor.options() 
    // way1: same to feats
    torch::Tensor feat_inpterp = torch::zeros({N, F}, feats.options());  

    // if specify other dtype and device
    // torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device()));

    // step1: grid and block size
    // 先明确需要多少个线程: N 和 F 的维度都可以并行

    // 两个维度需要平行运算! 第三个维度默认为1
    // 每个 block 的大小
    const dim3 block_size(16, 16);  // 128, 256, 512 需要多方尝试!

    // 计算出 grid 的大小: 一定要包含全部的 N 和 F
    const dim3 grid_size(
        (N + block_size.x - 1) / block_size.x, 
        (F + block_size.y - 1) / block_size.y
    );   // X 方向对应 N, 即不同的方块, Y 方向对应该方块中特征不同维度

    // TODO: 图解

    // 指定运算类型 && 启动 kernel 函数.
    AT_DISPATCH_FLOATING_TYPES(
        feats.type(),  // 对应的计算类型
        "trilinear_bw_cu", // 函数名, 报错名
        ([&] {
            trilinear_fw_kernel<scalar_t><<<grid_size, block_size>>>(  // kernel 的名字 <各种运算的数据类型&占位>
                // 对 torch::Tensor 需要将其转换为 CUDA 可见的
                // 将数据类型转型为 kernel (cuda) 可以识别的数据类型才可以被 kernel 函数处理 
                // scalar_t: 代替 float32, double, int 等, 或者一定是 float32 则可以直接用对应的数据类型即可
                // 3/2: 对应 tensor 的维度.
                // torch::RestrictPtrTraits: 所有元素不和其他元素有交集
                // size_t: idx 的形式 (int64)
                feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),  // input1
                points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),  // input2
                feat_inpterp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()  // output
                // 如果是普通的类型, lirujcjiw例如 bool, 则直接用即可
                // a,
            );
        }
        )
    );

    return feat_inpterp;
}


torch::Tensor trilinear_bw_cu(
    const torch::Tensor dL_dfeat_interp, 
    torch::Tensor feats, 
    torch::Tensor points 
) {

    // 需要有一个 output, first 定义一个空的, 然后不断填进去,初始先明确其形状.
    const int N = feats.size(0), F = feats.size(2);  // shape[0]

    // torch.zeros(N, F, dtype=torch.float32, device="cuda:0")
    // dtype, device all store in tensor.options() 
    // way1: same to feats
    torch::Tensor dl_dfeats = torch::zeros({N, 8, F}, feats.options());  

    // if specify other dtype and device
    // torch::zeros({N, F}, torch::dtype(torch::kInt32).device(feats.device()));

    // step1: grid and block size
    // 先明确需要多少个线程: N 和 F 的维度都可以并行

    // 两个维度需要平行运算! 第三个维度默认为1
    // 每个 block 的大小
    const dim3 block_size(16, 16);  // 128, 256, 512 需要多方尝试!

    // 计算出 grid 的大小: 一定要包含全部的 N 和 F
    const dim3 grid_size(
        (N + block_size.x - 1) / block_size.x, 
        (F + block_size.y - 1) / block_size.y
    );   // X 方向对应 N, 即不同的方块, Y 方向对应该方块中特征不同维度
    // TODO: 图解

    // 指定运算类型 && 启动 kernel 函数.
    AT_DISPATCH_FLOATING_TYPES(
        feats.type(),  // 对应的计算类型
        "trilinear_bw_cu", // 函数名, 报错名
        ([&] {
            trilinear_bw_kernel<scalar_t><<<grid_size, block_size>>>(  // kernel 的名字 <各种运算的数据类型&占位>
                // 对 torch::Tensor 需要将其转换为 CUDA 可见的
                // 将数据类型转型为 kernel (cuda) 可以识别的数据类型才可以被 kernel 函数处理 
                // scalar_t: 代替 float32, double, int 等, 或者一定是 float32 则可以直接用对应的数据类型即可
                // 3/2: 对应 tensor 的维度.
                // torch::RestrictPtrTraits: 所有元素不和其他元素有交集
                // size_t: idx 的形式 (int64)
                dL_dfeat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),  // input1
                feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),  // input1
                points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),  // input2
                dl_dfeats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>()  // output
                // 如果是普通的类型, lirujcjiw例如 bool, 则直接用即可
                // a,
            );
        }
        )
    );

    return dl_dfeats;
}

